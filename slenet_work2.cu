#include "hip/hip_runtime.h"
#include "slenet_params.h"
#include <stdio.h>
#include <stdlib.h>

#include <math.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hipblas.h>
#include <hip/hip_runtime_api.h>
//#include <hip/driver_types.h>

#define GEMM 1
#define CPU_GEMM 0 //0: GPU 1: CPU 
#define GEMM_GLOBAL 1

#define NUM_CLASSES 10

#define INPUT -1.0f // 1.0f //
#define WEIGHT -1.0f // 1.0f //
#define BIAS -1.0f // 1.0f //
#define CONV_POST_ACT 1.0f //  1.0f //
#define SS_POST_ACT  0.0f // 1.0f //
#define FC_POST_ACT  (1/(1+2.71828)) // 1.0f //

#define INSIZE 28
#define FILTER_SIZE 5
#define STRIDE 1
#define CHANNEL 6

#define CONV_OUTPUT_SIZE ((INSIZE - FILTER_SIZE)/STRIDE + 1) //24
#define SS_SIZE 4
#define SS_STRIDE 4
#define SS_CHANNELS 1

#define SS_OUTPUT_SIZE ((CONV_OUTPUT_SIZE - SS_SIZE)/SS_STRIDE + 1) //6

#define N1 CHANNEL*CONV_OUTPUT_SIZE*CONV_OUTPUT_SIZE
#define K1 64

#define N11 CONV_OUTPUT_SIZE*CONV_OUTPUT_SIZE
#define K11 64

#define N2 CHANNEL*SS_OUTPUT_SIZE*SS_OUTPUT_SIZE
#define K2 8

#define N3 NUM_CLASSES * SS_OUTPUT_SIZE * SS_OUTPUT_SIZE * SS_OUTPUT_SIZE
#define K3 10

#ifndef __DATE__
#define __DATE__
#endif

#ifndef __TIME__
#define __TIME__
#endif

#ifndef __FUNCTION__
#define __FUNCTION__
#endif

#ifndef __LINE__
#define __LINE__ 0
#endif

#ifndef __FILE__
#define __FILE__
#endif

int print_status = 1; 

void check_error(hipError_t status);
void check_error_extended(hipError_t status, const char *file, int line, const char *date_time);
void cublas_check_error_extended(hipblasStatus_t status, const char *file, int line, const char *date_time);
#define CHECK_CUDA(X) check_error_extended(X, __FILE__ " : " __FUNCTION__, __LINE__,  __DATE__ " - " __TIME__ );
#define CHECK_CUBLAS(X) cublas_check_error_extended(X, __FILE__ " : " __FUNCTION__, __LINE__,  __DATE__ " - " __TIME__ );

    hipblasHandle_t blas_handle();
    void free_pinned_memory();
    void pre_allocate_pinned_memory(size_t size);
    float *cuda_make_array_pinned_preallocated(float *x, size_t n);
    float *cuda_make_array_pinned(float *x, size_t n);
    float *cuda_make_array(float *x, size_t n);
    void **cuda_make_array_pointers(void **x, size_t n);
    int *cuda_make_int_array(size_t n);
	int *cuda_make_int_array_new_api(int *x, size_t n);
    void cuda_push_array(float *x_gpu, float *x, size_t n);
    //LIB_API void cuda_pull_array(float *x_gpu, float *x, size_t n);
    //LIB_API void cuda_set_device(int n);
    int cuda_get_device();
    void cuda_free_host(float *x_cpu);
    void cuda_free(float *x_gpu);
    void cuda_random(float *x_gpu, size_t n);
    float cuda_compare(float *x_gpu, float *x, size_t n, char *s);
    dim3 cuda_gridsize(size_t n);
    hipStream_t get_cuda_stream();
    //hipStream_t get_cuda_memcpy_stream();
    int get_number_of_blocks(int array_size, int block_size);
    int get_gpu_compute_capability(int i, char *device_name);
    void show_cuda_cudnn_info();

    hipStream_t switch_stream(int i);
    void wait_stream(int i);
    void reset_wait_stream_events();


int cuda_debug_sync = 0;

int cuda_get_device()
{
    int n = 0;
    hipError_t status = hipGetDevice(&n);
    CHECK_CUDA(status);
    return n;
}

void check_error(hipError_t status)
{
    hipError_t status2 = hipGetLastError();
    if (status != hipSuccess)
    {
        const char *s = hipGetErrorString(status);
        char buffer[256];
        printf("\n CUDA Error: %s\n", s);
        snprintf(buffer, 256, "CUDA Error: %s", s);
#ifdef WIN32
        getchar();
#endif
        //error(buffer, DARKNET_LOC); //jurn 
    }
    if (status2 != hipSuccess)
    {
        const char *s = hipGetErrorString(status2);
        char buffer[256];
        printf("\n CUDA Error Prev: %s\n", s);
        snprintf(buffer, 256, "CUDA Error Prev: %s", s);
#ifdef WIN32
        getchar();
#endif
        //error(buffer, DARKNET_LOC); //jurn 
    }
}

void check_error_extended(hipError_t status, const char *file, int line, const char *date_time)
{
    if (status != hipSuccess) {
        printf("CUDA status Error: file: %s() : line: %d : build time: %s \n", file, line, date_time);
        check_error(status);
    }
#if defined(DEBUG) || defined(CUDA_DEBUG)
    cuda_debug_sync = 1;
#endif
    if (cuda_debug_sync) {
        status = hipDeviceSynchronize();
        if (status != hipSuccess)
            printf("CUDA status = hipDeviceSynchronize() Error: file: %s() : line: %d : build time: %s \n", file, line, date_time);
    }
    check_error(status);
}



void cublas_check_error(hipblasStatus_t status)
{
#if defined(DEBUG) || defined(CUDA_DEBUG)
    hipDeviceSynchronize();
#endif
    if (cuda_debug_sync) {
        hipDeviceSynchronize();
    }
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("cuBLAS Error\n");
    }
}

void cublas_check_error_extended(hipblasStatus_t status, const char *file, int line, const char *date_time)
{
    if (status != HIPBLAS_STATUS_SUCCESS) {
      printf("\n cuBLAS status Error in: file: %s() : line: %d : build time: %s \n", file, line, date_time);
    }
#if defined(DEBUG) || defined(CUDA_DEBUG)
    cuda_debug_sync = 1;
#endif
    if (cuda_debug_sync) {
        hipError_t status = hipDeviceSynchronize();
      if (status != hipSuccess)
          printf("\n hipError_t status = hipDeviceSynchronize() Error in: file: %s() : line: %d : build time: %s \n", file, line, date_time);
    }
    cublas_check_error(status);
}

static hipStream_t streamsArray[16];    // hipStreamSynchronize( get_cuda_stream() );
static int streamInit[16] = { 0 };

hipStream_t get_cuda_stream() {
    int i = cuda_get_device();
    if (!streamInit[i]) {
        printf("Create CUDA-stream - %d \n", i);
#ifdef CUDNN
        hipError_t status = hipStreamCreateWithFlags(&streamsArray[i], hipStreamNonBlocking);
#else
        hipError_t status = hipStreamCreate(&streamsArray[i]);
#endif
        if (status != hipSuccess) {
            printf(" hipStreamCreate error: %d \n", status);
            const char *s = hipGetErrorString(status);
            printf("CUDA Error: %s\n", s);
            status = hipStreamCreateWithFlags(&streamsArray[i], hipStreamNonBlocking);    // hipStreamDefault
            CHECK_CUDA(status);
        }
        streamInit[i] = 1;
    }
    return streamsArray[i];
}


static int blasInit[16] = { 0 };
static hipblasHandle_t blasHandle[16];

hipblasHandle_t blas_handle()
{
    int i = cuda_get_device();
    if (!blasInit[i]) {
        CHECK_CUBLAS(hipblasCreate(&blasHandle[i]));
        hipblasStatus_t status = hipblasSetStream(blasHandle[i], get_cuda_stream());
        CHECK_CUBLAS(status);
        blasInit[i] = 1;
    }
    return blasHandle[i];
}

//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%

// Defining the mnist_data struct
typedef struct mnist_data {
	double data[INSIZE][INSIZE];
	unsigned int label;
} mnist_data;

// Defining the Layer class
class Layer {
	public:
		int M, N, O;
		float *pre_output, *output;
		float *weight, *bias;
    float *im2col_A; //for im2col
    float *gemm_B;
    float *gemm_C;

		Layer(int M, int N, int O);
		~Layer();

    void clear();
};

Layer::Layer(int M, int N, int O) {
	this->M = M;
	this->N = N;
	this->O = O;

	float *temp_weight, *temp_bias;

	// Initializing weights and biases
	temp_weight = (float*)malloc(sizeof(float) * M * N);
	temp_bias = (float*)malloc(sizeof(float) * N);

  temp_weight[0] = 0.0f; 
	for (int i = 0; i < M * N; i++){
		temp_weight[i] = WEIGHT; //1.0f;
    //temp_weight[i+1] = temp_weight[i]+1.0f; 
  }

	for (int i = 0; i < N; i++)
		temp_bias[i] = BIAS; //1.0f;

	// Allocating space for CUDA variables
	hipMalloc(&pre_output, sizeof(float) * O);
	hipMalloc(&output, sizeof(float) * O);
	hipMalloc(&weight, sizeof(float) * M * N);
	hipMalloc(&bias, sizeof(float) * N);

  hipMalloc(&im2col_A, sizeof(float) *M*O/N);
  hipMalloc(&gemm_B, sizeof(float) *M*N);
  hipMalloc(&gemm_C, sizeof(float) *(O/N)*N);


	// Copying weights and biases to CUDA variables
	hipMemcpy(weight, temp_weight, sizeof(float) * M * N, hipMemcpyHostToDevice);
	hipMemcpy(bias, temp_bias, sizeof(float) * N, hipMemcpyHostToDevice);

	// Freeing temporary weights and biases
	free(temp_weight);
	free(temp_bias);
}

Layer::~Layer() {
	// Freeing all CUDA varibles of a layer
	hipFree(pre_output);
	hipFree(output);
	hipFree(weight);
	hipFree(bias);
  hipFree(im2col_A);
}

void Layer::clear(){
	hipMemset(pre_output, 0x00, sizeof(float)*O); 
	hipMemset(output, 0x00, sizeof(float)*O); 
}

// Initializing a convolutional layer
Layer conv_layer(FILTER_SIZE * FILTER_SIZE, CHANNEL, CHANNEL * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE);
Layer ss_layer(SS_SIZE * SS_SIZE, SS_CHANNELS, CHANNEL * SS_OUTPUT_SIZE * SS_OUTPUT_SIZE);
Layer fc_layer(CHANNEL * SS_OUTPUT_SIZE * SS_OUTPUT_SIZE, NUM_CLASSES, NUM_CLASSES);

double time_taken = 0.0;

static unsigned int mnist_bin_to_int(unsigned char *tmp) {
	// Converting the binary char value to the integer value
	unsigned int result = 0;
	short charSize = 4;
	short multiplier = 256;

	for (int i = 0; i < charSize; i++) {
		unsigned int temp = tmp[i];

		for (int j = 0; j < charSize - i - 1; j++)
			temp *= multiplier;

		result += temp;
	}

	// Returning the integer value
	return result;
}

static int mnist_load(const char *image_filename, const char *label_filename, mnist_data **data_set, unsigned int *count) {
	// Initializing necessary variables
	FILE *images;
	FILE *labels;

	unsigned char *imagesBuffer;
	unsigned char *labelsBuffer;

	long imagesFileSize;
	long labelsFileSize;

	short unsignedIntSize = 4;
	short unsignedByteSize = 1;

	unsigned int imageMagicNumber;
	unsigned int labelMagicNumber;
	unsigned int imageTotalNumber;
	unsigned int labelTotalNumber;
	unsigned int rows, cols;

	// Opening image and label files of the test
	images = fopen("data/t10k-images.idx3-ubyte", "rb");

	if (images == NULL) {
		printf("Error! Images file cannot be read!\n");
		return 1;
	}

	labels = fopen("data/t10k-labels.idx1-ubyte", "rb");

	if (images == NULL) {
		printf("Error! Labels file cannot be read!\n");
		return 1;
	}

	fseek(images, 0, SEEK_END);
	fseek(labels, 0, SEEK_END);

	imagesFileSize = ftell(images);
	labelsFileSize = ftell(labels);

	fseek(images, 0, SEEK_SET);
	fseek(labels, 0, SEEK_SET);

	imagesBuffer = (unsigned char*)malloc(sizeof(unsigned char) * imagesFileSize);

	if (imagesBuffer == NULL) {
		printf("Error! Memory error has occured!\n");
		return 2;
	}

	labelsBuffer = (unsigned char*)malloc(sizeof(unsigned char) * labelsFileSize);

	if (labelsBuffer == NULL) {
		printf("Error! Memory error has occured!\n");
		return 2;
	}

	// Reading a magic number
	fread(imagesBuffer, unsignedIntSize, 1, images);
	fread(labelsBuffer, unsignedIntSize, 1, labels);
	imageMagicNumber = mnist_bin_to_int(imagesBuffer);
	labelMagicNumber = mnist_bin_to_int(labelsBuffer);
	printf("Image magic number: %d\n", imageMagicNumber);
	printf("Label magic number: %d\n", labelMagicNumber);

	// Reading a number of images and label files
	fread(imagesBuffer, unsignedIntSize, 1, images);
	fread(labelsBuffer, unsignedIntSize, 1, labels);
	imageTotalNumber = mnist_bin_to_int(imagesBuffer);
	labelTotalNumber = mnist_bin_to_int(labelsBuffer);
	printf("Number of images: %d\n", imageTotalNumber);
	printf("Number of labels: %d\n", labelTotalNumber);

	// Check whether the number of images and label files is the same
	if (imageTotalNumber != labelTotalNumber) {
		printf("Error! The number of images and the number of labels are different!\n");
		return 3;
	} else {
		printf("The number of images and the number of labels are the same!\n");
	}

	// Check the number of rows and columns
	fread(imagesBuffer, unsignedIntSize, 1, images);
	rows = mnist_bin_to_int(imagesBuffer);
	fread(imagesBuffer, unsignedIntSize, 1, images);
	cols = mnist_bin_to_int(imagesBuffer);
	printf("Rows: %d\n", rows);
	printf("Cols: %d\n", cols);

	*data_set = (mnist_data*)malloc(sizeof(mnist_data) * imageTotalNumber);

	// Load image data as double type
	for (int i = 0; i < imageTotalNumber; i++) {
		fread(imagesBuffer, rows * cols, 1, images);
		fread(labelsBuffer, unsignedByteSize, 1, labels);

		for (int j = 0; j < INSIZE; j++) {
			for (int k = 0; k < INSIZE; k++) {
				(*data_set)[i].data[j][k] = imagesBuffer[j * INSIZE + k] / 255.0;
			}
		}

		(*data_set)[i].label = labelsBuffer[0];
	}

	// Closing opened files
	fclose(images);
	fclose(labels);
	free(imagesBuffer);
	free(labelsBuffer);
	*count = imageTotalNumber;
	return 0;
}

// Printing MNIST data set examples
void printExamples(mnist_data **data_set, int count) {
	for (int i = 0; i < count; i++) {
		printf("\nImage:\n");

		for (int j = 0; j < INSIZE; j++) {
			for (int k = 0; k < INSIZE; k++) {
				if ((*data_set)[i].data[j][k] > 0) {
					printf("1");
				} else {
					printf("0");
				}
			}
			printf("\n");
		}

		printf("Label: %d\n", (*data_set)[i].label);
	}
}

__global__ void kernel_conv_filter(float input[INSIZE][INSIZE], float pre_output[CHANNEL][CONV_OUTPUT_SIZE][CONV_OUTPUT_SIZE], float weight[CHANNEL][FILTER_SIZE][FILTER_SIZE]) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int channel = idx % CHANNEL;
	int output_x = (idx / CHANNEL) % CONV_OUTPUT_SIZE;
	int output_y = (idx / CHANNEL / CONV_OUTPUT_SIZE) % CONV_OUTPUT_SIZE;
	float tempC = 0.0f;

	for (int i = 0; i < FILTER_SIZE; i++) {
		for (int j = 0; j < FILTER_SIZE; j++) {
			tempC += weight[channel][i][j] * input[i + output_x][j + output_y];
		}
	}
  if (idx < CHANNEL*CONV_OUTPUT_SIZE*CONV_OUTPUT_SIZE)
	  pre_output[channel][output_x][output_y] = tempC;
}


__global__ void kernel_conv_bias(float pre_output[CHANNEL][CONV_OUTPUT_SIZE][CONV_OUTPUT_SIZE], float bias[CHANNEL]) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int channel = idx % CHANNEL;
	int output_x = (idx / CHANNEL) % CONV_OUTPUT_SIZE;
	int output_y = (idx / CHANNEL / CONV_OUTPUT_SIZE) % CONV_OUTPUT_SIZE;
  if (idx < CHANNEL*CONV_OUTPUT_SIZE*CONV_OUTPUT_SIZE)
	  pre_output[channel][output_x][output_y] += bias[channel];
}

__global__ void kernel_conv_sigmoid(float preact[CHANNEL][CONV_OUTPUT_SIZE][CONV_OUTPUT_SIZE], float output[CHANNEL][CONV_OUTPUT_SIZE][CONV_OUTPUT_SIZE]) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int channel = idx % CHANNEL;
	int output_x = (idx / CHANNEL) % CONV_OUTPUT_SIZE;
	int output_y = (idx / CHANNEL / CONV_OUTPUT_SIZE) % CONV_OUTPUT_SIZE;
  if (idx < CHANNEL*CONV_OUTPUT_SIZE*CONV_OUTPUT_SIZE)
	  output[channel][output_x][output_y] = 1 / (1 + exp(-preact[channel][output_x][output_y]));
}

__global__ void kernel_ss1_filter(float input[CHANNEL][CONV_OUTPUT_SIZE][CONV_OUTPUT_SIZE], float pre_output[CHANNEL][SS_OUTPUT_SIZE][SS_OUTPUT_SIZE], float weight[SS_CHANNELS][SS_SIZE][SS_SIZE]) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int channel = idx % CHANNEL;
	int output_x = (idx / CHANNEL) % SS_OUTPUT_SIZE;
	int output_y = (idx / CHANNEL / SS_OUTPUT_SIZE) % SS_OUTPUT_SIZE;
	float tempC = 0.0f;

	for (int i = 0; i < SS_SIZE; i++) {
		for (int j = 0; j < SS_SIZE; j++) {
			tempC += weight[0][i][j] * input[channel][i + output_x * SS_STRIDE][j + output_y * SS_STRIDE];
		}
	}
  if (idx < CHANNEL*SS_OUTPUT_SIZE*SS_OUTPUT_SIZE)
	  pre_output[channel][output_x][output_y] = tempC;
}

__global__ void kernel_ss1_bias(float pre_output[CHANNEL][SS_OUTPUT_SIZE][SS_OUTPUT_SIZE], float bias[SS_CHANNELS]) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int channel = idx % CHANNEL;
	int output_x = (idx / CHANNEL) % SS_OUTPUT_SIZE;
	int output_y = (idx / CHANNEL / SS_OUTPUT_SIZE) % SS_OUTPUT_SIZE;
  if (idx < CHANNEL*SS_OUTPUT_SIZE*SS_OUTPUT_SIZE)
	  pre_output[channel][output_x][output_y] += bias[0];
}

__global__ void kernel_ss1_sigmoid(float pre_output[CHANNEL][SS_OUTPUT_SIZE][SS_OUTPUT_SIZE], float output[CHANNEL][SS_OUTPUT_SIZE][SS_OUTPUT_SIZE]) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int channel = idx % CHANNEL;
	int output_x = (idx / CHANNEL) % SS_OUTPUT_SIZE;
	int output_y = (idx / CHANNEL / SS_OUTPUT_SIZE) % SS_OUTPUT_SIZE;
  if (idx < CHANNEL*SS_OUTPUT_SIZE*SS_OUTPUT_SIZE)
	  output[channel][output_x][output_y] = 1 / (1 + exp(-pre_output[channel][output_x][output_y]));
}

__global__ void kernel_fc1(float input[CHANNEL][SS_OUTPUT_SIZE][SS_OUTPUT_SIZE], float pre_output[NUM_CLASSES], 
                            float weight[NUM_CLASSES][CHANNEL][SS_OUTPUT_SIZE][SS_OUTPUT_SIZE]) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int channel = idx % NUM_CLASSES;
	float tempC = 0.0f;

	for (int i = 0; i < CHANNEL; i++) {
		for (int j = 0; j < SS_OUTPUT_SIZE; j++) {
			for (int k = 0; k < SS_OUTPUT_SIZE; k++) {
				tempC += weight[channel][i][j][k] * input[i][j][k];
			}
		}
	}

	pre_output[channel] = tempC;
}

__global__ void kernel_fc1_bias(float pre_output[NUM_CLASSES], float bias[NUM_CLASSES]) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int channel = idx % NUM_CLASSES;
	pre_output[channel] += bias[channel];
}

__global__ void kernel_fc1_sigmoid(float pre_output[NUM_CLASSES], float output[NUM_CLASSES]) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int channel = idx % NUM_CLASSES;
	output[channel] = 1 / (1 + exp(-pre_output[channel]));
}

//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
__global__ void ker2col_kernel(float weight_col[CHANNEL][FILTER_SIZE*FILTER_SIZE], float weight[CHANNEL][FILTER_SIZE][FILTER_SIZE]) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x; 
  
  int channel = idx % CHANNEL;
	int x = (idx / CHANNEL) % FILTER_SIZE;
	int y = (idx / CHANNEL / FILTER_SIZE) % FILTER_SIZE;
  if (idx < CHANNEL*FILTER_SIZE*FILTER_SIZE)
	  weight_col[channel][x*FILTER_SIZE + y] = weight[channel][x][y];
}

__global__ void gemm_global_kernel(float matB[CHANNEL][FILTER_SIZE*FILTER_SIZE] 
                      ,float matA[FILTER_SIZE*FILTER_SIZE][CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE]
                      ,float matC[CHANNEL][CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE]) {

  int idx = threadIdx.x + blockIdx.x * blockDim.x; 

  int x = idx % CHANNEL;
	int y = (idx / CHANNEL) % (CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE);

  matC[x][y] = 0.0f; 
  for(int i=0; i<FILTER_SIZE*FILTER_SIZE; i++){
    matC[x][y] += matB[x][i] * matA[i][y]; 
  } 
 
}

__global__ void col2im_kernel(float preout[CHANNEL][CONV_OUTPUT_SIZE][CONV_OUTPUT_SIZE], float preout_col[CONV_OUTPUT_SIZE*CONV_OUTPUT_SIZE][CHANNEL]) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x; 
  
  int channel = idx % CHANNEL;
	int x = (idx / CHANNEL) % CONV_OUTPUT_SIZE;
	int y = (idx / CHANNEL / CONV_OUTPUT_SIZE) % CONV_OUTPUT_SIZE;
  if (idx < CHANNEL*CONV_OUTPUT_SIZE*CONV_OUTPUT_SIZE)
	  preout[channel][x][y] = preout_col[x*FILTER_SIZE + y][channel];
}

void verifyConv(float *A, float val) {
	float maxError = 0.0f;

  int cnt = 0; 
	for (int i = 0; i < CHANNEL * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE; i++){ 
		maxError = max(abs(A[i] - val), maxError);
    if (maxError != 0)
      cnt++; 
  }
	printf("maxError = %f (cnt = %d),%d)\n", maxError, cnt, CHANNEL*CONV_OUTPUT_SIZE*CONV_OUTPUT_SIZE);
}


void verifyim2col(float *A, float val) {
	float maxError = 0.0f;

  int cnt = 0; 
	for (int i = 0; i < FILTER_SIZE * FILTER_SIZE * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE; i++){ 
		maxError = max(abs(A[i] - val), maxError);
    if (maxError != 0)
      cnt++; 
  }
	printf("maxError = %f (cnt = %d),%d)\n", maxError, cnt, FILTER_SIZE * FILTER_SIZE *CONV_OUTPUT_SIZE*CONV_OUTPUT_SIZE);
}


void verify_gemmB(float *A, float val) {
	float maxError = 0.0f;

  int cnt = 0; 
	for (int i = 0; i < CHANNEL * FILTER_SIZE * FILTER_SIZE; i++){ 
		maxError = max(abs(A[i] - val), maxError);
    if (maxError != 0)
      cnt++; 
  }
	printf("maxError = %f (cnt = %d),%d)\n", maxError, cnt, CHANNEL*FILTER_SIZE*FILTER_SIZE);
}

void verifySS(float *A, float val) {
	float maxError = 0.0f;

  int cnt = 0;
	for (int i = 0; i < CHANNEL * SS_OUTPUT_SIZE * SS_OUTPUT_SIZE; i++){
		maxError = max(abs(A[i] - val), maxError);
    if (maxError > 0.0001f)
      cnt++; 
  }
	printf("maxError = %f (cnt = %d),%d)\n", maxError, cnt, CHANNEL*SS_OUTPUT_SIZE*SS_OUTPUT_SIZE);
}

void verifyFC(float *A, float val) {
	float maxError = 0.0f;

  int cnt = 0;
	for (int i = 0; i < NUM_CLASSES; i++){
		maxError = max(abs(A[i] - val), maxError);
    if (maxError > 0.0009f)
      cnt++; 
  }
	printf("maxError = %f (cnt = %d),%d)\n", maxError, cnt, NUM_CLASSES);
}

// CUDA: grid stride looping
#define CUDA_KERNEL_LOOP(i, n) \
   for (int i = blockIdx.x * blockDim.x + threadIdx.x; \
        i < (n); \
        i += blockDim.x * gridDim.x)

// https://github.com/BVLC/caffe/blob/master/src/caffe/util/im2col.cu
__global__ void im2col_gpu_kernel(const int n, const float* data_im,
        const int height, const int width, const int ksize,
        const int pad,
        const int stride,
        const int height_col, const int width_col,
        float *data_col) {

    int index = blockIdx.x*blockDim.x+threadIdx.x;

    for(; index < n; index += blockDim.x*gridDim.x){
        int w_out = index % width_col;
        int h_index = index / width_col;
        int h_out = h_index % height_col;
        int channel_in = h_index / height_col;
        int channel_out = channel_in * ksize * ksize;
        int h_in = h_out * stride - pad;
        int w_in = w_out * stride - pad;
        float* data_col_ptr = data_col;
        data_col_ptr += (channel_out * height_col + h_out) * width_col + w_out;
        const float* data_im_ptr = data_im;
        data_im_ptr += (channel_in * height + h_in) * width + w_in;
        for (int i = 0; i < ksize; ++i) {
            for (int j = 0; j < ksize; ++j) {
                int h = h_in + i;
                int w = w_in + j;

                *data_col_ptr = (h >= 0 && w >= 0 && h < height && w < width) ?
                    data_im_ptr[i * width + j] : 0;

                //data_im[(channel_in * height + h_in) * width + w_in + i * width + j];
                //(*data_col_ptr) = data_im_ptr[ii * width + jj];

                data_col_ptr += height_col * width_col;
            }
        }
    }
}

///*
void gemm_ongpu(int TA, int TB, int M, int N, int K, float ALPHA,
        float *A_gpu, int lda,
        float *B_gpu, int ldb,
        float BETA,
        float *C_gpu, int ldc)
{
    hipblasHandle_t handle = blas_handle();
    //hipError_t stream_status = (hipError_t)hipblasSetStream(handle, get_cuda_stream());
    //CHECK_CUDA(stream_status);
    //hipError_t status = (hipError_t)hipblasSgemm(handle, (TB ? HIPBLAS_OP_T : HIPBLAS_OP_N),
    //        (TA ? HIPBLAS_OP_T : HIPBLAS_OP_N), N, M, K, &ALPHA, B_gpu, ldb, A_gpu, lda, &BETA, C_gpu, ldc);
    //CHECK_CUDA(status);
    hipError_t status = (hipError_t)hipblasSgemm(handle, (TB ? HIPBLAS_OP_T : HIPBLAS_OP_N),
            (TA ? HIPBLAS_OP_T : HIPBLAS_OP_N), N, M, K, &ALPHA, B_gpu, ldb, A_gpu, lda, &BETA, C_gpu, ldc);
}
//*/





// Performing a forward pass using a single image
static double forward_pass(double data[INSIZE][INSIZE], bool verify) {
	// Copying a double data to a float data
	float input[INSIZE][INSIZE];
	float *verification;
  
  input[0][0] = 0.0f; 
	for (int i = 0; i < INSIZE; i++) {
		for (int j = 0; j < INSIZE; j++){
      //input[i][j] = INPUT; //Simulated data
			input[i][j] = data[i][j];  //MNIST data
      //printf("%f ",input[i][j]);
    }
    //printf("\n");
	}

//Layer conv_layer(FILTER_SIZE * FILTER_SIZE, CHANNEL, CHANNEL * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE);
//Layer ss_layer(SS_SIZE * SS_SIZE, SS_CHANNELS, CHANNEL * SS_OUTPUT_SIZE * SS_OUTPUT_SIZE);
//Layer fc_layer(CHANNEL * SS_OUTPUT_SIZE * SS_OUTPUT_SIZE, NUM_CLASSES, NUM_CLASSES);

  //l_input.clear(); 
	//conv_layer.clear(); 
	//ss_layer.clear();
	//fc_layer.clear();

	float (*d_input)[INSIZE];
	hipMalloc(&d_input, sizeof(float) * INSIZE * INSIZE);
	hipMemcpy(d_input, input, sizeof(float) * INSIZE * INSIZE, hipMemcpyHostToDevice);

  //For im2col workspace 
  //float* im2col_input;  
  //hipMalloc(&im2col_input, sizeof(float) * INSIZE * INSIZE);

  //float* im2col_workspace;  
  //hipMalloc(&im2col_workspace, sizeof(float) * FILTER_SIZE * FILTER_SIZE * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);


float matB[CHANNEL][FILTER_SIZE * FILTER_SIZE];
float matA[FILTER_SIZE * FILTER_SIZE][CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE];
float matC[CHANNEL][CONV_OUTPUT_SIZE*CONV_OUTPUT_SIZE];
float newMatC[CHANNEL][CONV_OUTPUT_SIZE][CONV_OUTPUT_SIZE];
float gemmMatC[CHANNEL][CONV_OUTPUT_SIZE][CONV_OUTPUT_SIZE];

	// Verifying im2col operation
	if (verify) { //verify
    if (print_status == 1){
		printf("Verifying im2col_A: ");
		verification = (float*)malloc(sizeof(float) * INSIZE * INSIZE );
		hipMemcpy(verification, d_input, sizeof(float) * INSIZE *INSIZE, hipMemcpyDeviceToHost);
		//verifyConv(verification, INPUT); //-1.0f 

    #if 1
    for(int i=0; i<INSIZE*INSIZE; i++){ 
      
      if (i%(INSIZE) == 0){
           printf("\n");
      }
      printf("%2.1f ", verification[i]);
    }
    printf("\n");
    #endif 
    free(verification);
    print_status--;
    }
  }  


#if GEMM // gemm or direct setting 
  //im2col_gpu_kernel_ext<<<(N1+K1-1)/K1, K1>>>(CONV_OUTPUT_SIZE*CONV_OUTPUT_SIZE, d_input, INSIZE, INSIZE, FILTER_SIZE, FILTER_SIZE, 0, 0, STRIDE, STRIDE, 1, 1, CONV_OUTPUT_SIZE, CONV_OUTPUT_SIZE,ic_workspace);
///*
  im2col_gpu_kernel<<<(N11+K11-1)/K11, K11>>>(CONV_OUTPUT_SIZE*CONV_OUTPUT_SIZE, //num_kernels, = channels * height_col * width_col; 
                                              (float *)d_input, //data_im, 
                                              INSIZE, //height, 
                                              INSIZE, //width, 
                                              FILTER_SIZE, //ksize, 
                                              0, //pad,
                                              STRIDE, //stride, 
                                              CONV_OUTPUT_SIZE, //height_col, 
                                              CONV_OUTPUT_SIZE, //width_col, 
                                              (float *)conv_layer.im2col_A); //data_col);
                                      
//*/


///*
if (verify) { //verify
  if (print_status == 1){    
    printf("Verifying im2col_A: ");
		verification = (float*)malloc(sizeof(float) * FILTER_SIZE * FILTER_SIZE * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE);
		hipMemcpy(verification, conv_layer.im2col_A, sizeof(float) * FILTER_SIZE * FILTER_SIZE * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE, hipMemcpyDeviceToHost);
		verifyim2col(verification, INPUT); //-1.0f 
    free(verification);

  #if 1
      for(int i=0; i<FILTER_SIZE*FILTER_SIZE*CONV_OUTPUT_SIZE*CONV_OUTPUT_SIZE; i++){ 
      //FILTER_SIZE * FILTER_SIZE
      if (i%(CONV_OUTPUT_SIZE*CONV_OUTPUT_SIZE) == 0){
           printf("\n");
      }
      printf("%3.1f ", verification[i]);
    }
    printf("\n");
    
  #endif 
    print_status--;
    }
  }   
//*/

///*
  //float matA[FILTER_SIZE * FILTER_SIZE][CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE];
  hipMemcpy(matA, conv_layer.im2col_A, sizeof(float) * FILTER_SIZE * FILTER_SIZE * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE, hipMemcpyDeviceToHost);
  #if 0
    if (print_status == 1){
      for(int i=0; i<FILTER_SIZE * FILTER_SIZE; i++){
        for(int j=0; j<CONV_OUTPUT_SIZE*CONV_OUTPUT_SIZE; j++){
          if(j% CONV_OUTPUT_SIZE == 0)
            printf("\n");
          printf("%3.2f ", matA[i][j]);
        }   
        printf("\n");
    }
    printf("\n\n");
    print_status--; 
    }
  #endif 
//*/
		
  //ker2col operation 
  ker2col_kernel<<<CHANNEL, FILTER_SIZE * FILTER_SIZE>>>((float(*)[FILTER_SIZE*FILTER_SIZE])conv_layer.gemm_B , (float(*)[FILTER_SIZE][FILTER_SIZE])conv_layer.weight); 

	// Verifying ker2col operation
	if (verify) { //verify
    if (print_status == 1){
		printf("Veri gemmB ker2col: ");
		verification = (float*)malloc(sizeof(float) * FILTER_SIZE * FILTER_SIZE * CHANNEL);
		hipMemcpy(verification, conv_layer.weight, sizeof(float) * FILTER_SIZE * FILTER_SIZE * CHANNEL, hipMemcpyDeviceToHost);
		verify_gemmB(verification, WEIGHT); //-1.0f

  #if 1 //conv_layer.weight
      for(int i=0; i<FILTER_SIZE * FILTER_SIZE*CHANNEL; i++){ 
      
      //FILTER_SIZE * FILTER_SIZE
      if (i%(FILTER_SIZE * FILTER_SIZE) == 0){
           printf("\n");
      }
      printf("%3.1f ", verification[i]);
    }
  #endif 
    free(verification);
    print_status--; 
    }
  }

  //float matB[CHANNEL][FILTER_SIZE * FILTER_SIZE]; 
  hipMemcpy(matB, conv_layer.gemm_B, sizeof(float) * FILTER_SIZE * FILTER_SIZE * CHANNEL, hipMemcpyDeviceToHost);

  #if 0  //conv_layer.gemm_B
  if (print_status == 1){
  for(int i=0; i<CHANNEL; i++){
    for(int j=0; j<FILTER_SIZE * FILTER_SIZE; j++){ 
      if(j% FILTER_SIZE == 0)
            printf("\n");
          printf("%3.2f ", matB[i][j]);
    }
    printf("\n");
  }
  printf("\n");
  print_status--; 
  }
  #endif 

///* //on cpu gemm 	
#if CPU_GEMM 
#if 1
  //float matC[CHANNEL][CONV_OUTPUT_SIZE*CONV_OUTPUT_SIZE]; 
  //gemm_custom_cpu(); 
  {
    //matC memset 
    for(int i=0; i<CHANNEL ; i++){
      for(int j=0; j< CONV_OUTPUT_SIZE*CONV_OUTPUT_SIZE ; j++){
        for(int k=0; k< FILTER_SIZE*FILTER_SIZE; k++){
          matC[i][j] =0.0f; 
        }
      }
    }

    for(int i=0; i<CHANNEL ; i++){
      for(int j=0; j< CONV_OUTPUT_SIZE*CONV_OUTPUT_SIZE ; j++){
        for(int k=0; k< FILTER_SIZE*FILTER_SIZE; k++){
          matC[i][j] += matB[i][k] * matA[k][j]; 
        }
      }
    }
  }
  #if 0 //matC: print debug 
  {
    for(int i=0; i< ; i++){ //CHANNEL
      for(int j=0; j< CONV_OUTPUT_SIZE*CONV_OUTPUT_SIZE ; j++){
          if (j%CONV_OUTPUT_SIZE == 0)
            printf("\n");
          printf("%3.2f ", matC[i][j]); 

      }
      printf("\n");
    }
    printf("\n");
  }
  #endif 
#endif 


//transform matC[6][576] to newMatC[6][24][24]

    int k=0; 
    for(int i=0; i<CHANNEL ; i++){
      for(int j=0; j< CONV_OUTPUT_SIZE*CONV_OUTPUT_SIZE ; j++){
          if (j%CONV_OUTPUT_SIZE == 0){
            int k = j/CONV_OUTPUT_SIZE;
            //printf("\n"); 
          }
          newMatC[i][j/CONV_OUTPUT_SIZE][j%CONV_OUTPUT_SIZE] = matC[i][j];
          //printf("%3.0f ", newMatC[i][k][j]); 

      }
      //printf("\n");
    }

// print newMatC
  #if 0
    if (print_status == 1){
      for(int i=0; i<CHANNEL; i++){ //
        for(int j=0; j<CONV_OUTPUT_SIZE; j++){
          for(int k=0; k<CONV_OUTPUT_SIZE ; k++){
            printf("%3.2f ", newMatC[i][j][k]);
          }
          printf("\n");
        }   
        printf("\n");
    }
    printf("\n\n");
    print_status--; 
    }
  #endif


	//kernel_conv_filter<<<(N1+K1-1)/K1, K1>>>(d_input, 
  //                                          (float(*)[CONV_OUTPUT_SIZE][CONV_OUTPUT_SIZE])conv_layer.pre_output,
  //                                          (float(*)[FILTER_SIZE][FILTER_SIZE])conv_layer.weight);

  hipMemcpy(conv_layer.pre_output, matC, sizeof(float) * CHANNEL * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE, hipMemcpyHostToDevice);
  //hipMemcpy(conv_layer.pre_output, newMatC, sizeof(float) * CHANNEL * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE, hipMemcpyHostToDevice); //both are okay


  // Verifying Convolutional filtering operation on CPU
	if (verify) {
		printf("Veri Convolutional filtering (for gemm): ");
		verification = (float*)malloc(sizeof(float) * CHANNEL * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE);
		hipMemcpy(verification, conv_layer.pre_output, sizeof(float) * CHANNEL * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE, hipMemcpyDeviceToHost);
		verifyConv(verification, INPUT*WEIGHT*FILTER_SIZE*FILTER_SIZE); //25.0f
		free(verification);
    printf("\n");
	}

  #if 0
  if (print_status == 1){
  hipMemcpy(gemmMatC, conv_layer.pre_output, sizeof(float) * CHANNEL * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE, hipMemcpyDeviceToHost);
  
      for(int i=0; i<1; i++){ //CHANNEL
        for(int j=0; j<CONV_OUTPUT_SIZE; j++){
          for(int k=0; k<CONV_OUTPUT_SIZE ; k++){
            printf("%3.2f ", gemmMatC[i][j][k]);
          }
          printf("\n");
        }   
        printf("\n");
    }
    print_status--; 
    printf("\n\n");
  }
  #endif 
//*/

#elif GEMM_GLOBAL// GPU_GEMM 

  //sgmm global operation  <<<CHANNEL, CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE>>>
  //dim3 numBlocks()
  gemm_global_kernel<<<CHANNEL, CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE>>>((float(*)[FILTER_SIZE*FILTER_SIZE])conv_layer.gemm_B 
                      ,(float(*)[CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE])conv_layer.im2col_A
                      ,(float(*)[CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE])conv_layer.gemm_C); 

  // Verifying Convolutional filtering operation on GPU
	if (verify) { //verify
		printf("Veri Convolutional filtering (for gemm GPU global): ");
		verification = (float*)malloc(sizeof(float) * CHANNEL * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE);
		hipMemcpy(verification, conv_layer.pre_output, sizeof(float) * CHANNEL * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE, hipMemcpyDeviceToHost);
		verifyConv(verification, INPUT*WEIGHT*FILTER_SIZE*FILTER_SIZE); //25.0f
		free(verification);
    printf("\n");
	}

  #if 0
  if (print_status == 1){
  //hipMemcpy(gemmMatC, conv_layer.pre_output, sizeof(float) * CHANNEL * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE, hipMemcpyDeviceToHost);
  hipMemcpy(gemmMatC, conv_layer.gemm_C, sizeof(float) * CHANNEL * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE, hipMemcpyDeviceToHost);
  
      for(int i=0; i<1; i++){ //CHANNEL
        for(int j=0; j<CONV_OUTPUT_SIZE; j++){
          for(int k=0; k<CONV_OUTPUT_SIZE ; k++){
            printf("%3.2f ", gemmMatC[i][j][k]);
          }
          printf("\n");
        }   
        printf("\n");
    }
    print_status--; 
    printf("\n\n");
  }
  #endif 

    //col2im operation 
  col2im_kernel<<<(N1+K1-1)/K1, K1>>>((float(*)[CONV_OUTPUT_SIZE][CONV_OUTPUT_SIZE])conv_layer.pre_output, 
                                      (float(*)[CHANNEL])conv_layer.gemm_C); 


	// Verifying col2im operation
	if (verify) {
		printf("Verifying col2im: ");
		verification = (float*)malloc(sizeof(float) * CHANNEL * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE);
		hipMemcpy(verification, conv_layer.pre_output, sizeof(float) * CHANNEL * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE, hipMemcpyDeviceToHost);
		verifyConv(verification, INPUT*WEIGHT*FILTER_SIZE*FILTER_SIZE); //25.0f
		free(verification);
	}

  #if 1
  if (print_status == 1){
  hipMemcpy(gemmMatC, conv_layer.pre_output, sizeof(float) * CHANNEL * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE, hipMemcpyDeviceToHost);
  //hipMemcpy(gemmMatC, conv_layer.gemm_C, sizeof(float) * CHANNEL * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE, hipMemcpyDeviceToHost);
  
      for(int i=0; i<1; i++){ //CHANNEL
        for(int j=0; j<CONV_OUTPUT_SIZE; j++){
          for(int k=0; k<CONV_OUTPUT_SIZE ; k++){
            printf("%3.2f ", gemmMatC[i][j][k]);
          }
          printf("\n");
        }   
        printf("\n");
    }
    print_status--; 
    printf("\n\n");
  }
  #endif  

#else
  int n = CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE; //l.out_w*l.out_h
  int k = FILTER_SIZE * FILTER_SIZE; // l.size*l.size
  int m = CHANNEL; // l.n / l.groups

  //For gemm workspace 
  //float *a = (float*)malloc(sizeof(float) * M * N);

  float *a = conv_layer.gemm_B; //l.weights_gpu + j*l.nweights / l.groups;
  float *b = conv_layer.im2col_A; //state.workspace
  float *c = conv_layer.gemm_C; //l.output_gpu + (i*l.groups + j)*n*m;

  gemm_ongpu(0, 1, m, n, k, 1, a, k, b, k, 1, c, n);

	// Verifying hipblasSgemm operation
	if (verify) {
		printf("Veri hipblasSgemm: ");
		verification = (float*)malloc(sizeof(float) * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE * CHANNEL);
		hipMemcpy(verification, conv_layer.gemm_C, sizeof(float) * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE * CHANNEL, hipMemcpyDeviceToHost);
		verifyConv(verification, INPUT*WEIGHT*FILTER_SIZE*FILTER_SIZE); //25.0f
		free(verification);
	}

  #if 1
  if (print_status == 1){
    verification = (float*)malloc(sizeof(float) * CHANNEL * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE);
    hipMemcpy(verification, fc_layer.output, sizeof(float) * CHANNEL * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE, hipMemcpyDeviceToHost);
  
      for(int i=0; i<CHANNEL * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE; i++){
        if (i % CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE == 0){
          printf("\n");
          
        }
        printf("%3.1f ", verification[i]);
      }
      printf("\n");

    print_status--; 
    
    printf("\n");
    free(verification);
  }
  #endif 


  #if 0
  if (print_status == 1){
  hipMemcpy(newMatC, conv_layer.pre_output, sizeof(float) * CHANNEL * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE, hipMemcpyDeviceToHost);
  
      for(int i=0; i<CHANNEL; i++){
        for(int j=0; j<CONV_OUTPUT_SIZE; j++){
          for(int k=0; k<CONV_OUTPUT_SIZE ; k++){
            printf("%3.2f ", newMatC[i][j][k]);
          }
          printf("\n");
        }   
        printf("\n");
    }
    print_status--; 
    printf("\n\n");
  }
  #endif 



#endif 
  //%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%

#else
	// Performing Convolutional filtering
	kernel_conv_filter<<<(N1+K1-1)/K1, K1>>>(d_input, 
                                            (float(*)[CONV_OUTPUT_SIZE][CONV_OUTPUT_SIZE])conv_layer.pre_output,
                                            (float(*)[FILTER_SIZE][FILTER_SIZE])conv_layer.weight);
  
	// Verifying Convolutional filtering operation
	if (verify) {
		printf("Veri Convolutional filtering: ");
		verification = (float*)malloc(sizeof(float) * CHANNEL * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE);
		hipMemcpy(verification, conv_layer.pre_output, sizeof(float) * CHANNEL * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE, hipMemcpyDeviceToHost);
		verifyConv(verification, INPUT*WEIGHT*FILTER_SIZE*FILTER_SIZE); //25.0f
		free(verification);
	}

  #if 0
  if (print_status == 1){
  hipMemcpy(newMatC, conv_layer.pre_output, sizeof(float) * CHANNEL * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE, hipMemcpyDeviceToHost);
  
      for(int i=0; i<CHANNEL; i++){
        for(int j=0; j<CONV_OUTPUT_SIZE; j++){
          for(int k=0; k<CONV_OUTPUT_SIZE ; k++){
            printf("%3.2f ", newMatC[i][j][k]);
          }
          printf("\n");
        }   
        printf("\n");
    }
    print_status--; 
    printf("\n\n");
  }
  #endif 
#endif 

	// Performing Convolutional bias addition
	kernel_conv_bias<<<(N1+K1-1)/K1, K1>>>((float(*)[CONV_OUTPUT_SIZE][CONV_OUTPUT_SIZE])conv_layer.pre_output, conv_layer.bias);

	// Verifying Convolutional bias operation
	if (verify) {
		printf("Veri Convolutional bias: ");
		verification = (float*)malloc(sizeof(float) * CHANNEL * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE);
		hipMemcpy(verification, conv_layer.pre_output, sizeof(float) * CHANNEL * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE, hipMemcpyDeviceToHost);
		verifyConv(verification, INPUT*WEIGHT*FILTER_SIZE*FILTER_SIZE + BIAS); // 26.0f
		free(verification);
	}

	// Performing Convolutional sigmoid operation
	kernel_conv_sigmoid<<<(N1+K1-1)/K1, K1>>>((float(*)[CONV_OUTPUT_SIZE][CONV_OUTPUT_SIZE])conv_layer.pre_output, (float(*)[CONV_OUTPUT_SIZE][CONV_OUTPUT_SIZE])conv_layer.output);

	// Verifying Convolutional sigmoid operation
	if (verify) {
		printf("Veri Convolutional sigmoid: ");
		verification = (float*)malloc(sizeof(float) * CHANNEL * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE);
		hipMemcpy(verification, conv_layer.output, sizeof(float) * CHANNEL * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE, hipMemcpyDeviceToHost);
		verifyConv(verification, CONV_POST_ACT); // 1.0f
		free(verification);
	}

	// Performing Subsampling filtering
	kernel_ss1_filter<<<(N2+K2-1)/K2, K2>>>((float(*)[CONV_OUTPUT_SIZE][CONV_OUTPUT_SIZE])conv_layer.output, (float(*)[SS_OUTPUT_SIZE][SS_OUTPUT_SIZE])ss_layer.pre_output, (float(*)[SS_SIZE][SS_SIZE])ss_layer.weight);

	// Verifying Subsampling filtering operation
	if (verify) {
		printf("Veri Subsampling filtering: ");
		verification = (float*)malloc(sizeof(float) * CHANNEL * SS_OUTPUT_SIZE * SS_OUTPUT_SIZE);
		hipMemcpy(verification, ss_layer.pre_output, sizeof(float) * CHANNEL * SS_OUTPUT_SIZE * SS_OUTPUT_SIZE, hipMemcpyDeviceToHost);
		verifySS(verification, CONV_POST_ACT*WEIGHT*SS_SIZE*SS_SIZE); //16.0f
		free(verification);
	}

	// Performing Subsampling bias addition
	kernel_ss1_bias<<<(N2+K2-1)/K2, K2>>>((float(*)[SS_OUTPUT_SIZE][SS_OUTPUT_SIZE])ss_layer.pre_output, ss_layer.bias);

	// Verifying Subsampling bias operation
	if (verify) {
		printf("Veri Subsampling bias: ");
		verification = (float*)malloc(sizeof(float) * CHANNEL * SS_OUTPUT_SIZE * SS_OUTPUT_SIZE);
		hipMemcpy(verification, ss_layer.pre_output, sizeof(float) * CHANNEL * SS_OUTPUT_SIZE * SS_OUTPUT_SIZE, hipMemcpyDeviceToHost);
		verifySS(verification, CONV_POST_ACT*WEIGHT*SS_SIZE*SS_SIZE + BIAS); // 17.0f
		free(verification);
	}

	// // Performing Subsampling sigmoid operation
	kernel_ss1_sigmoid<<<(N2+K2-1)/K2, K2>>>((float(*)[SS_OUTPUT_SIZE][SS_OUTPUT_SIZE])ss_layer.pre_output, (float(*)[SS_OUTPUT_SIZE][SS_OUTPUT_SIZE])ss_layer.output);

	// Verifying Subsampling sigmoid operation
	if (verify) {
		printf("Veri Subsampling sigmoid: ");
		verification = (float*)malloc(sizeof(float) * CHANNEL * SS_OUTPUT_SIZE * SS_OUTPUT_SIZE);
		hipMemcpy(verification, ss_layer.output, sizeof(float) * CHANNEL * SS_OUTPUT_SIZE * SS_OUTPUT_SIZE, hipMemcpyDeviceToHost);
		verifySS(verification, SS_POST_ACT); //1.0f
		free(verification);
	}

	// Performing Fully-Connected Computation
	kernel_fc1<<<(N3+K3-1)/K3, K3>>>((float(*)[SS_OUTPUT_SIZE][SS_OUTPUT_SIZE])ss_layer.output, (float(*))fc_layer.pre_output, (float(*)[CHANNEL][SS_OUTPUT_SIZE][SS_OUTPUT_SIZE])fc_layer.weight);

	// Verifying Fully-Connected Computation
	if (verify) {
		printf("Veri Fully-Connected: ");
		verification = (float*)malloc(sizeof(float) * NUM_CLASSES);
		hipMemcpy(verification, fc_layer.pre_output, sizeof(float) * NUM_CLASSES, hipMemcpyDeviceToHost);
		verifyFC(verification, SS_POST_ACT*WEIGHT*CHANNEL*SS_OUTPUT_SIZE*SS_OUTPUT_SIZE);//216.0f
		free(verification);
	}

	// Performing Fully-Connected bias operation
	kernel_fc1_bias<<<1, K3>>>((float(*))fc_layer.pre_output, fc_layer.bias);

	// Verifying Fully-Connected bias operation
	if (verify) {
		printf("Veri Fully-Connected bias: ");
		verification = (float*)malloc(sizeof(float) * NUM_CLASSES);
		hipMemcpy(verification, fc_layer.pre_output, sizeof(float) * NUM_CLASSES, hipMemcpyDeviceToHost);
		verifyFC(verification, SS_POST_ACT*WEIGHT*CHANNEL*SS_OUTPUT_SIZE*SS_OUTPUT_SIZE + BIAS); //217.0f
		free(verification);
	}

	// Performing Fully-Connected sigmoid operation
	kernel_fc1_sigmoid<<<1, K3>>>((float(*))fc_layer.pre_output, (float(*))fc_layer.output);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	// Verifying Fully-Connected sigmoid operation
	if (verify) {
		printf("Veri Fully-Connected sigmoid: ");
		verification = (float*)malloc(sizeof(float) * NUM_CLASSES);
		hipMemcpy(verification, fc_layer.output, sizeof(float) * NUM_CLASSES, hipMemcpyDeviceToHost);
		verifyFC(verification, FC_POST_ACT); // 1.0f
		free(verification);
	}

  #if 1
  if (print_status == 1 || print_status == 0 || print_status == -1 || print_status == -2 || print_status == -3 ){
    verification = (float*)malloc(sizeof(float) * NUM_CLASSES);
    hipMemcpy(verification, fc_layer.output, sizeof(float) * NUM_CLASSES, hipMemcpyDeviceToHost);
  
      for(int i=0; i<NUM_CLASSES; i++){
        printf("%3.1f[%d] ", verification[i], i);
    }

    print_status--; 
    
    printf("\n");
    free(verification);
  }
  #endif 

	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);

	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipFree(d_input);
	return elapsedTime;
}

void copy_trained_parameters() {
	hipMemcpy(conv_layer.weight, c1_weight, sizeof(float) * CHANNEL * FILTER_SIZE * FILTER_SIZE, hipMemcpyHostToDevice);
	hipMemcpy(conv_layer.bias, c1_bias, sizeof(float) * CHANNEL, hipMemcpyHostToDevice);
	hipMemcpy(ss_layer.weight, s2_weight, sizeof(float) * SS_SIZE * SS_SIZE, hipMemcpyHostToDevice);
	hipMemcpy(ss_layer.bias, s2_bias, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(fc_layer.weight, f3_weight, sizeof(float) * NUM_CLASSES * SS_OUTPUT_SIZE * SS_OUTPUT_SIZE * SS_OUTPUT_SIZE, hipMemcpyHostToDevice);
	hipMemcpy(fc_layer.bias, f3_bias, sizeof(float) * NUM_CLASSES, hipMemcpyHostToDevice);
}

int main() {
	int ret, i;
	mnist_data *test_set;
	static unsigned int test_cnt;

	// Calling the mnist_load() function
	if (ret = mnist_load("data/t10k-images.idx3-ubyte", "data/t10k-labels.idx1-ubyte", &test_set, &test_cnt) != 0) {
		printf("An error occured: %d\n", ret);
	} else {
		printf("test_cnt = %d\n", test_cnt);
	}

	// Verifying the image and label data of the specified number of examples
	//printExamples(&test_set, 1);

	// Verifying the convolutional layer
	double data[INSIZE][INSIZE];

  data[0][0] = 0.0f; 
	for (i = 0; i < INSIZE; i++) {
		for (int j = 0; j < INSIZE; j++){
			data[i][j] = INPUT; //1.0f;
      //data[i][j+1] =  data[i][j] + 1.0f; 
      //printf("%3.0f ", data[i][j]);
    }
    //printf("\n");
	}

  //copy_trained_parameters();

	//forward_pass(data, true);

	copy_trained_parameters();

	// Performing forward pass
	unsigned int error = 0;
	unsigned int max = 0;
	float res[10];

	for (i = 0; i < test_cnt; i++) { //test_cnt
		time_taken += forward_pass(test_set[i].data, false);
		hipMemcpy(res, fc_layer.output, sizeof(float) * NUM_CLASSES, hipMemcpyDeviceToHost);

		for (int j = 0; j < NUM_CLASSES; j++) {
			if (res[max] < res[j])
				max = j;
		}

		if (max != test_set[i].label)
			error++;
	}

	printf("Error Rate = %f%% (%d out of 10000)\n", double(error) / double(test_cnt) * 100.0, error);
	printf("Accuracy = %.3f%% (%d out of 10000)\n", 100.0 - double(error) / double(test_cnt) * 100.0, test_cnt - error);
	printf("Execution time = %f (ms) \n", time_taken);

	free(test_set);
	return 0;
}
